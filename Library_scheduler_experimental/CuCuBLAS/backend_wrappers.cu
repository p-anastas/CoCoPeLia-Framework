///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief The DGEMM CoCopeLia implementation.
///

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <cblas.h>

#include "backend_lib_wrappers.hpp"
#include "backend_wrappers.hpp"

hipblasHandle_t handle[128] = {NULL};

void backend_init(short dev_id, CQueue_p h2d_q, CQueue_p d2h_q, CQueue_p exec_q){
  int dev_idc = -1;
  hipError_t err = hipGetDevice(&dev_idc);
  massert(dev_idc == dev_id,
    "backend_init: called on different device - actual(%d) vs called(%d)\n", dev_idc, dev_id);
  massert(HIPBLAS_STATUS_SUCCESS == hipblasCreate(&(handle[dev_id])), "hipblasCreate failed\n");
  massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(handle[dev_id],
    *(hipStream_t*) (exec_q->cqueue_backend_ptr)), "hipblasSetStream failed\n");
  return;
}

int CoCoPeLiaGetDevice(){
  int dev_id = -1;
  hipError_t err = hipGetDevice(&dev_id);
  massert(hipSuccess == err,
    "CoCoPeLiaGetDevice: hipGetDevice failed - %s\n", hipGetErrorString(err));
  return dev_id;
}

void CoCoPeLiaSelectDevice(short dev_id){
  hipSetDevice(dev_id);
  hipError_t err = hipSetDevice(dev_id);
  massert(hipSuccess == err,
    "CoCoPeLiaSelectDevice: hipSetDevice failed - %s\n", hipGetErrorString(err));
}
void CoCoPeLiaDevGetMemInfo(long long* free_dev_mem, long long* max_dev_mem){
  size_t free_dev_mem_tmp, max_dev_mem_tmp;
    hipError_t err = hipMemGetInfo(&free_dev_mem_tmp, &max_dev_mem_tmp);
  	massert(hipSuccess == err,
      "CoCoPeLiaDevGetMemInfo: hipMemGetInfo failed - %s\n", hipGetErrorString(err));
    *free_dev_mem = (long long) free_dev_mem_tmp;
    *max_dev_mem = (long long) max_dev_mem_tmp;
}

void backend_run_operation(short dev_id, void* backend_data, const char* opname){
  short lvl = 5;
  if (!strcmp(opname, "gemm")){
    gemm_backend_in_p ptr_ker_translate = (gemm_backend_in_p) backend_data;
  #ifdef DDEBUG
  	lprintf(lvl, "backend_run_operation: hipblasDgemm(handle[%d], TransA = %c, TransB = %c, M = %d, N = %d, K = %d, alpha = %lf, A = %p, lda = %d, \n\
  	B = %p, ldb = %d, beta = %lf, C = %p, ldC = %d)\n", dev_id, ptr_ker_translate->TransA, ptr_ker_translate->TransB,
  		ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, ptr_ker_translate->alpha, (VALUE_TYPE*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
  		(VALUE_TYPE*) *ptr_ker_translate->B, ptr_ker_translate->ldB, ptr_ker_translate->beta, (VALUE_TYPE*) *ptr_ker_translate->C, ptr_ker_translate->ldC);
  #endif
  	massert(HIPBLAS_STATUS_SUCCESS == hipblasDgemm(handle[dev_id], OpCharToCublas(ptr_ker_translate->TransA), OpCharToCublas(ptr_ker_translate->TransB),
  		ptr_ker_translate->M, ptr_ker_translate->N, ptr_ker_translate->K, &ptr_ker_translate->alpha, (VALUE_TYPE*) *ptr_ker_translate->A, ptr_ker_translate->ldA,
  		(VALUE_TYPE*) *ptr_ker_translate->B, ptr_ker_translate->ldB, &ptr_ker_translate->beta, (VALUE_TYPE*) *ptr_ker_translate->C, ptr_ker_translate->ldC),
  		"backend_run_operation: hipblasDgemm failed\n");
  }
  else error("backend_run_operation: unkown opname=%s\n", opname);
}
