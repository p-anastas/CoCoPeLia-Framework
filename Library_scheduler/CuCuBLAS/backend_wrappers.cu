///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief The DGEMM CoCopeLia implementation.
///

#include <cblas.h>

#include "backend_lib_wrappers.hpp"
#include "unihelpers.hpp"

int CoCoPeLiaGetDevice(){
  int dev_id = -1;
  hipError_t err = hipGetDevice(&dev_id);
  massert(hipSuccess == err,
    "CoCoPeLiaGetDevice: hipGetDevice failed - %s\n", hipGetErrorString(err));
  return dev_id;
}

void CoCoPeLiaSelectDevice(short dev_id){
  hipSetDevice(dev_id);
  hipError_t err = hipSetDevice(dev_id);
  massert(hipSuccess == err,
    "CoCoPeLiaSelectDevice: hipSetDevice failed - %s\n", hipGetErrorString(err));
}
void CoCoPeLiaDevGetMemInfo(long long* free_dev_mem, long long* max_dev_mem){
  size_t free_dev_mem_tmp, max_dev_mem_tmp;
    hipError_t err = hipMemGetInfo(&free_dev_mem_tmp, &max_dev_mem_tmp);
  	massert(hipSuccess == err,
      "CoCoPeLiaDevGetMemInfo: hipMemGetInfo failed - %s\n", hipGetErrorString(err));
    *free_dev_mem = (long long) free_dev_mem_tmp;
    *max_dev_mem = (long long) max_dev_mem_tmp;
}
