///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some CUDA function calls with added error-checking
///

#include <cstdio>
#include <typeinfo>
#include <float.h>
#include <hiprand.h>

#include "backend_wrappers.hpp"

/*****************************************************/
/// Command queue class functions
CommandQueue::CommandQueue()
{
	cqueue_backend_ptr = malloc(sizeof(hipStream_t));
	hipError_t err = hipStreamCreate((hipStream_t*) cqueue_backend_ptr);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - %s\n", hipGetErrorString(err));
}

void CommandQueue::sync_barrier()
{
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipStreamSynchronize(stream);
	massert(hipSuccess == err, "CommandQueue::sync_barrier - %s\n", hipGetErrorString(err));
}

void CommandQueue::wait_for_event(Event_p Wevent)
{
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipEvent_t cuda_event= *(hipEvent_t*) Wevent->event_backend_ptr;
	hipError_t err = hipStreamWaitEvent(stream, cuda_event, 0); // 0-only parameter = future NVIDIA masterplan?
	massert(hipSuccess == err, "CommandQueue::wait_for_event - %s\n", hipGetErrorString(err));
}

/*****************************************************/
/// Event class functions
Event::Event()
{
	event_backend_ptr = malloc(sizeof(hipEvent_t));
	hipError_t err = hipEventCreate(( hipEvent_t*) event_backend_ptr);
	massert(hipSuccess == err, "Event::Event - %s\n", hipGetErrorString(err));
}

void Event::sync_barrier()
{
	hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
	hipError_t err = hipEventSynchronize(cuda_event);
	massert(hipSuccess == err, "Event::sync_barrier - %s\n", hipGetErrorString(err));
}

void Event::record_to_queue(CQueue_p Rr){
	hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
	hipStream_t stream = *((hipStream_t*) Rr->cqueue_backend_ptr);
	hipError_t err = hipEventRecord(cuda_event, stream);
	massert(hipSuccess == err, "Event::record_to_queue - %s\n", hipGetErrorString(err));
}

short Event::is_complete(){
	hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
	hipError_t err = hipEventQuery(cuda_event);
	if (err == hipSuccess) return 1;
	if (err == hipErrorNotReady) return 0;
	else error("Event::is_complete - %s\n", hipGetErrorString(err));
}

/*****************************************************/
/// Event-based timer class functions

Event_timer::Event_timer() {
  Event_start = new Event();
  Event_stop = new Event();
  time_ms = 0;
}

void Event_timer::start_point(CQueue_p start_queue)
{
	Event_start->record_to_queue(start_queue);
	//hipStream_t stream = *((hipStream_t*) start_queue->cqueue_backend_ptr);
	//hipEvent_t cuda_event = *(hipEvent_t*) Event_start->event_backend_ptr;
	//hipEventRecord(cuda_event, stream);
}

void Event_timer::stop_point(CQueue_p stop_queue)
{
	Event_stop->record_to_queue(stop_queue);
	//hipStream_t stream = *((hipStream_t*) stop_queue->cqueue_backend_ptr);
	//hipEvent_t cuda_event = *(hipEvent_t*) Event_stop->event_backend_ptr;
	//hipEventRecord(cuda_event, stream);
}

double Event_timer::sync_get_time()
{
	float temp_t;
	//hipEvent_t cuda_event_start = *(hipEvent_t*) Event_start->event_backend_ptr;
	//hipEvent_t cuda_event_stop = *(hipEvent_t*) Event_stop->event_backend_ptr;
	//hipEventSynchronize(cuda_event_start);
	//hipEventSynchronize(cuda_event_stop);
	Event_start->sync_barrier();
	Event_stop->sync_barrier();
	hipEvent_t cuda_event_start = *(hipEvent_t*) Event_start->event_backend_ptr;
	hipEvent_t cuda_event_stop = *(hipEvent_t*) Event_stop->event_backend_ptr;
	hipEventElapsedTime(&temp_t, cuda_event_start, cuda_event_stop);
	time_ms = (double) temp_t;
	return time_ms;
}

/*****************************************************/
