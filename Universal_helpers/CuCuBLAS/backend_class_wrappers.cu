///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some CUDA function calls with added error-checking
///

#include <cstdio>
#include <typeinfo>
#include <float.h>
#include <hiprand.h>

#include "backend_wrappers.hpp"

int Event_num_device[128] = {0};
#ifndef UNIHELPER_LOCKFREE_ENABLE
int unihelper_lock = 0;
#endif

inline void get_lock(){
#ifndef UNIHELPER_LOCKFREE_ENABLE
	while(__sync_lock_test_and_set (&unihelper_lock, 1));
#endif
	;
}
inline void release_lock(){
#ifndef UNIHELPER_LOCKFREE_ENABLE
	__sync_lock_release(&unihelper_lock);
#endif
	;
}

/*****************************************************/
/// Event Status-related functions

const char* print_event_status(event_status in_status){
	switch(in_status){
		case(UNRECORDED):
			return "UNRECORDED";
		case(RECORDED):
			return "RECORDED";
		case(COMPLETE):
			return "COMPLETE";
		case(CHECKED):
			return "CHECKED";
		case(GHOST):
			return "GHOST";
		default:
			error("print_event_status: Unknown state\n");
	}
}

/*****************************************************/
/// Command queue class functions
CommandQueue::CommandQueue(int dev_id_in)
{
	int prev_dev_id = CoCoPeLiaGetDevice();
	dev_id = dev_id_in;
	CoCoPeLiaSelectDevice(dev_id);
#ifdef ENABLE_PARALLEL_BACKEND
	backend_ctr = 0;
	for (int par_idx = 0; par_idx < MAX_BACKEND_L; par_idx++ ){
		cqueue_backend_ptr[par_idx] = malloc(sizeof(hipStream_t));
		hipError_t err = hipStreamCreate((hipStream_t*) cqueue_backend_ptr[par_idx]);
		massert(hipSuccess == err, "CommandQueue::CommandQueue(%d) - %s\n", dev_id, hipGetErrorString(err));
		hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[par_idx]);

		cqueue_backend_data[par_idx] = malloc(sizeof(hipblasHandle_t));
		massert(HIPBLAS_STATUS_SUCCESS == hipblasCreate((hipblasHandle_t*) cqueue_backend_data[par_idx]),
			"CommandQueue::CommandQueue(%d): hipblasCreate failed\n", dev_id);
		massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(*((hipblasHandle_t*) cqueue_backend_data[par_idx]), stream),
			"CommandQueue::CommandQueue(%d): hipblasSetStream failed\n", dev_id);
	}
#else
	cqueue_backend_ptr = malloc(sizeof(hipStream_t));
	hipError_t err = hipStreamCreate((hipStream_t*) cqueue_backend_ptr);
	massert(hipSuccess == err, "CommandQueue::CommandQueue(%d) - %s\n", dev_id, hipGetErrorString(err));
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);

	cqueue_backend_data = malloc(sizeof(hipblasHandle_t));
	massert(HIPBLAS_STATUS_SUCCESS == hipblasCreate((hipblasHandle_t*) cqueue_backend_data),
		"CommandQueue::CommandQueue(%d): hipblasCreate failed\n", dev_id);
	massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(*((hipblasHandle_t*) cqueue_backend_data), stream),
		"CommandQueue::CommandQueue(%d): hipblasSetStream failed\n", dev_id);
#endif
	CoCoPeLiaSelectDevice(prev_dev_id);
}

CommandQueue::~CommandQueue()
{
#ifdef ENABLE_PARALLEL_BACKEND
	for (int par_idx = 0; par_idx < MAX_BACKEND_L; par_idx++ ){
		hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[par_idx]);
		hipError_t err = hipStreamSynchronize(stream);
		massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamSynchronize: %s\n", hipGetErrorString(err));
		err = hipStreamDestroy(stream);
		massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamDestroy: %s\n", hipGetErrorString(err));
		free(cqueue_backend_ptr[par_idx]);
		hipblasHandle_t handle = *((hipblasHandle_t*) cqueue_backend_data[par_idx]);
		massert(HIPBLAS_STATUS_SUCCESS == hipblasDestroy(handle),
			"CommandQueue::CommandQueue - hipblasDestroy(handle) failed\n");
	}
#else
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipStreamSynchronize(stream);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamSynchronize: %s\n", hipGetErrorString(err));
	err = hipStreamDestroy(stream);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamDestroy: %s\n", hipGetErrorString(err));
	free(cqueue_backend_ptr);
	hipblasHandle_t handle = *((hipblasHandle_t*) cqueue_backend_data);
	massert(HIPBLAS_STATUS_SUCCESS == hipblasDestroy(handle),
		"CommandQueue::CommandQueue - hipblasDestroy(handle) failed\n");
#endif
	return;
}

void CommandQueue::sync_barrier()
{
#ifdef ENABLE_PARALLEL_BACKEND
	for (int par_idx = 0; par_idx < MAX_BACKEND_L; par_idx++ ){
		hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[par_idx]);
		hipError_t err = hipStreamSynchronize(stream);
		massert(hipSuccess == err, "CommandQueue::sync_barrier - %s\n", hipGetErrorString(err));
	}
#else
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipStreamSynchronize(stream);
	massert(hipSuccess == err, "CommandQueue::sync_barrier - %s\n", hipGetErrorString(err));
#endif
}

void CommandQueue::add_host_func(void* func, void* data){
	get_lock();
#ifdef ENABLE_PARALLEL_BACKEND
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[backend_ctr]);
	hipError_t err = hipLaunchHostFunc(stream, (hipHostFn_t) func, data);
	massert(hipSuccess == err, "CommandQueue::add_host_func - %s\n", hipGetErrorString(err));
#else
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipLaunchHostFunc(stream, (hipHostFn_t) func, data);
	massert(hipSuccess == err, "CommandQueue::add_host_func - %s\n", hipGetErrorString(err));
#endif
	release_lock();
}

void CommandQueue::wait_for_event(Event_p Wevent)
{
	if (Wevent->query_status() == CHECKED) return;
	get_lock();
#ifdef ENABLE_PARALLEL_BACKEND
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr[backend_ctr]);
	hipEvent_t cuda_event= *(hipEvent_t*) Wevent->event_backend_ptr;
	hipError_t err = hipStreamWaitEvent(stream, cuda_event, 0); // 0-only parameter = future NVIDIA masterplan?
	massert(hipSuccess == err, "CommandQueue::wait_for_event - %s\n", hipGetErrorString(err));
#else
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipEvent_t cuda_event= *(hipEvent_t*) Wevent->event_backend_ptr;
	hipError_t err = hipStreamWaitEvent(stream, cuda_event, 0); // 0-only parameter = future NVIDIA masterplan?
	massert(hipSuccess == err, "CommandQueue::wait_for_event - %s\n", hipGetErrorString(err));
#endif
	release_lock();
}

#ifdef ENABLE_PARALLEL_BACKEND
void CommandQueue::request_parallel_backend()
{
	get_lock();
	if (backend_ctr == MAX_BACKEND_L - 1) backend_ctr = 0;
	else backend_ctr++;
	release_lock();
}
#endif

/*****************************************************/
/// Event class functions. TODO: Do status = .. commands need lock?
Event::Event(int dev_id_in)
{
	get_lock();
	event_backend_ptr = malloc(sizeof(hipEvent_t));
	dev_id = dev_id_in;
	Event_num_device[idxize(dev_id)]++;
	id = Event_num_device[idxize(dev_id)];
	hipError_t err = hipEventCreate(( hipEvent_t*) event_backend_ptr);
	status = UNRECORDED;
	massert(hipSuccess == err, "Event::Event - %s\n", hipGetErrorString(err));
	release_lock();
}

Event::~Event()
{
	sync_barrier();
	get_lock();
	Event_num_device[idxize(dev_id)]--;
	hipError_t err = hipEventDestroy(*(( hipEvent_t*) event_backend_ptr));
	free(event_backend_ptr);
	massert(hipSuccess == err, "Event::~Event - %s\n", hipGetErrorString(err));
	release_lock();
}

void Event::sync_barrier()
{
	get_lock();
	if (status != CHECKED){
		if (status == UNRECORDED){
			warning("Event::sync_barrier: Tried to sync unrecorded event\n");
			return;
		}
		hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
		hipError_t err = hipEventSynchronize(cuda_event);
		if (status == RECORDED) status = CHECKED;
		massert(hipSuccess == err, "Event::sync_barrier - %s\n", hipGetErrorString(err));
	}
	release_lock();
}

void Event::record_to_queue(CQueue_p Rr){
	get_lock();
	if (Rr == NULL) status = CHECKED;
	else{
		if (status != UNRECORDED){
			warning("Event(%d,dev_id = %d)::record_to_queue(%d): Recording %s event\n", id, dev_id, Rr->dev_id, print_event_status(status));
		}
#ifdef ENABLE_PARALLEL_BACKEND
		hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
		hipStream_t stream = *((hipStream_t*) Rr->cqueue_backend_ptr[Rr->backend_ctr]);
		hipError_t err = hipEventRecord(cuda_event, stream);
#else
		hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
		hipStream_t stream = *((hipStream_t*) Rr->cqueue_backend_ptr);
		hipError_t err = hipEventRecord(cuda_event, stream);
#endif
		status = RECORDED;
		massert(hipSuccess == err, "Event(%d,dev_id = %d)::record_to_queue(%d) - %s\n",  id, dev_id, Rr->dev_id, hipGetErrorString(err));
	}
	release_lock();
}

event_status Event::query_status(){
	get_lock();
	enum event_status local_status = status;
	if (local_status != CHECKED){
		hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
		hipError_t err = hipEventQuery(cuda_event);

		if (err == hipSuccess && (local_status == UNRECORDED ||  local_status == COMPLETE));
		else if (err == hipSuccess && local_status == RECORDED) local_status = status = COMPLETE;
		else if (err == hipErrorNotReady && local_status == RECORDED);
		else if (err == hipErrorNotReady && local_status == UNRECORDED){
#ifdef DEBUG
			// this should not happen in a healthy locked update scenario.
			warning("Event::query_status: hipErrorNotReady with status == UNRECORDED should not happen\n");
#endif
			local_status = status = RECORDED;
		}
		else if (err == hipSuccess &&  local_status == CHECKED){
			;
			// TODO: This should not happen in a healthy locked update scenario.
			// But it does since no locking yet. Not sure of its effects.
#ifdef DEBUG
			warning("Event::query_status: hipSuccess with local_status == CHECKED should not happen\n");
#endif
		}
		else error("Event::query_status - %s, local_status=%s, status = %s\n",
		hipGetErrorString(err), print_event_status(local_status), print_event_status(status));
	}
	release_lock();
	return local_status;
}

void Event::checked(){
	get_lock();
	if (status == COMPLETE) status = CHECKED;
	else error("Event::checked(): error event was %s,  not COMPLETE()\n", print_event_status(status));
	release_lock();
}

void Event::reset(){
	get_lock();
	status = UNRECORDED;
	release_lock();
}

/*****************************************************/
/// Event-based timer class functions

Event_timer::Event_timer(int dev_id) {
  Event_start = new Event(dev_id);
  Event_stop = new Event(dev_id);
  time_ms = 0;
}

void Event_timer::start_point(CQueue_p start_queue)
{
	Event_start->record_to_queue(start_queue);
}

void Event_timer::stop_point(CQueue_p stop_queue)
{
	Event_stop->record_to_queue(stop_queue);
}

double Event_timer::sync_get_time()
{
	float temp_t;
	Event_start->sync_barrier();
	Event_stop->sync_barrier();
	hipEvent_t cuda_event_start = *(hipEvent_t*) Event_start->event_backend_ptr;
	hipEvent_t cuda_event_stop = *(hipEvent_t*) Event_stop->event_backend_ptr;
	hipEventElapsedTime(&temp_t, cuda_event_start, cuda_event_stop);
	time_ms = (double) temp_t;
	return time_ms;
}

/*****************************************************/
