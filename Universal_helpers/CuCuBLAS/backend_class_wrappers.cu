///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some CUDA function calls with added error-checking
///

#include <cstdio>
#include <typeinfo>
#include <float.h>
#include <hiprand.h>

#include "backend_wrappers.hpp"

int Event_num_device[128] = {0};
#ifndef UNIHELPER_LOCKFREE_ENABLE
int unihelper_lock = 0;
#endif

inline void get_lock(){
#ifndef UNIHELPER_LOCKFREE_ENABLE
	while(__sync_lock_test_and_set (&unihelper_lock, 1));
#endif
	;
}
inline void release_lock(){
#ifndef UNIHELPER_LOCKFREE_ENABLE
	__sync_lock_release(&unihelper_lock);
#endif
	;
}

/*****************************************************/
/// Event Status-related functions

const char* print_event_status(event_status in_status){
	switch(in_status){
		case(UNRECORDED):
			return "UNRECORDED";
		case(RECORDED):
			return "RECORDED";
		case(COMPLETE):
			return "COMPLETE";
		case(CHECKED):
			return "CHECKED";
		case(GHOST):
			return "GHOST";
		default:
			error("print_event_status: Unknown state\n");
	}
}

/*****************************************************/
/// Command queue class functions
CommandQueue::CommandQueue()
{
	cqueue_backend_ptr = malloc(sizeof(hipStream_t));
	hipError_t err = hipStreamCreate((hipStream_t*) cqueue_backend_ptr);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - %s\n", hipGetErrorString(err));
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);

	cqueue_backend_data = malloc(sizeof(hipblasHandle_t));
	massert(HIPBLAS_STATUS_SUCCESS == hipblasCreate((hipblasHandle_t*) cqueue_backend_data),
		"CommandQueue::CommandQueue: hipblasCreate failed\n");
	massert(HIPBLAS_STATUS_SUCCESS == hipblasSetStream(*((hipblasHandle_t*) cqueue_backend_data), stream),
		"hipblasSetStream failed\n");
}

CommandQueue::~CommandQueue()
{
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipStreamSynchronize(stream);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamSynchronize: %s\n", hipGetErrorString(err));
	err = hipStreamDestroy(stream);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - hipStreamDestroy: %s\n", hipGetErrorString(err));
	free(cqueue_backend_ptr);
	hipblasHandle_t handle = *((hipblasHandle_t*) cqueue_backend_data);
	massert(HIPBLAS_STATUS_SUCCESS == hipblasDestroy(handle),
		"CommandQueue::CommandQueue - hipblasDestroy(handle) failed\n");
	return;
}

void CommandQueue::sync_barrier()
{
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipStreamSynchronize(stream);
	massert(hipSuccess == err, "CommandQueue::sync_barrier - %s\n", hipGetErrorString(err));
}

void CommandQueue::add_host_func(void* func, void* data){
	get_lock();
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipLaunchHostFunc(stream, (hipHostFn_t) func, data);
	massert(hipSuccess == err, "CommandQueue::add_host_func - %s\n", hipGetErrorString(err));
	release_lock();
}

void CommandQueue::wait_for_event(Event_p Wevent)
{
	if (Wevent->query_status() == CHECKED) return;
	get_lock();
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipEvent_t cuda_event= *(hipEvent_t*) Wevent->event_backend_ptr;
	hipError_t err = hipStreamWaitEvent(stream, cuda_event, 0); // 0-only parameter = future NVIDIA masterplan?
	massert(hipSuccess == err, "CommandQueue::wait_for_event - %s\n", hipGetErrorString(err));
	release_lock();
}

/*****************************************************/
/// Event class functions. TODO: Do status = .. commands need lock?
Event::Event()
{
	get_lock();
	event_backend_ptr = malloc(sizeof(hipEvent_t));
	int dev_id;  hipGetDevice(&dev_id);
	Event_num_device[dev_id]++;
	id = Event_num_device[dev_id];
	hipError_t err = hipEventCreate(( hipEvent_t*) event_backend_ptr);
	status = UNRECORDED;
	massert(hipSuccess == err, "Event::Event - %s\n", hipGetErrorString(err));
	release_lock();
}

Event::~Event()
{
	sync_barrier();
	get_lock();
	int dev_id;  hipGetDevice(&dev_id);
	Event_num_device[dev_id]--;
	hipError_t err = hipEventDestroy(*(( hipEvent_t*) event_backend_ptr));
	free(event_backend_ptr);
	massert(hipSuccess == err, "Event::~Event - %s\n", hipGetErrorString(err));
	release_lock();
}

void Event::sync_barrier()
{
	get_lock();
	if (status != CHECKED){
		if (status == UNRECORDED){
			warning("Event::sync_barrier: Tried to sync unrecorded event\n");
			return;
		}
		hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
		hipError_t err = hipEventSynchronize(cuda_event);
		if (status == RECORDED) status = CHECKED;
		massert(hipSuccess == err, "Event::sync_barrier - %s\n", hipGetErrorString(err));
	}
	release_lock();
}

void Event::record_to_queue(CQueue_p Rr){
	get_lock();
	if (Rr == NULL) status = CHECKED;
	else{
		if (status != UNRECORDED){
			warning("Event::record_to_queue: Recording %s event\n", print_event_status(status));
		}
		hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
		hipStream_t stream = *((hipStream_t*) Rr->cqueue_backend_ptr);
		hipError_t err = hipEventRecord(cuda_event, stream);
		status = RECORDED;
		massert(hipSuccess == err, "Event::record_to_queue - %s\n", hipGetErrorString(err));
	}
	release_lock();
}

event_status Event::query_status(){
	get_lock();
	if (status != CHECKED){
		hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
		hipError_t err = hipEventQuery(cuda_event);

		if (err == hipSuccess && (status == UNRECORDED ||  status == COMPLETE));
		else if (err == hipSuccess && status == RECORDED) status = COMPLETE;
		else if (err == hipErrorNotReady && status == RECORDED);
		else if (err == hipErrorNotReady && status == UNRECORDED){
			// this should not happen in a healthy implementation
			warning("Event::query_status: hipErrorNotReady with status == UNRECORDED should not happen\n");
			status = RECORDED;
		}
		else if (err == hipSuccess &&  status == CHECKED)
			// TODO: This should not happen in a healthy locked update scenario.
			// But it does since no locking yet. Not sure of its effects.
			warning("Event::query_status: hipSuccess with status == CHECKED should not happen\n");
		else error("Event::query_status - %s, status=%s\n", hipGetErrorString(err), print_event_status(status));
	}
	release_lock();
	return status;
}

void Event::checked(){
	get_lock();
	if (status == COMPLETE) status = CHECKED;
	else error("Event::checked(): error event was %s,  not COMPLETE()\n", print_event_status(status));
	release_lock();
}

void Event::reset(){
	get_lock();
	status = UNRECORDED;
	release_lock();
}

/*****************************************************/
/// Event-based timer class functions

Event_timer::Event_timer() {
  Event_start = new Event();
  Event_stop = new Event();
  time_ms = 0;
}

void Event_timer::start_point(CQueue_p start_queue)
{
	Event_start->record_to_queue(start_queue);
	//hipStream_t stream = *((hipStream_t*) start_queue->cqueue_backend_ptr);
	//hipEvent_t cuda_event = *(hipEvent_t*) Event_start->event_backend_ptr;
	//hipEventRecord(cuda_event, stream);
}

void Event_timer::stop_point(CQueue_p stop_queue)
{
	Event_stop->record_to_queue(stop_queue);
	//hipStream_t stream = *((hipStream_t*) stop_queue->cqueue_backend_ptr);
	//hipEvent_t cuda_event = *(hipEvent_t*) Event_stop->event_backend_ptr;
	//hipEventRecord(cuda_event, stream);
}

double Event_timer::sync_get_time()
{
	float temp_t;
	//hipEvent_t cuda_event_start = *(hipEvent_t*) Event_start->event_backend_ptr;
	//hipEvent_t cuda_event_stop = *(hipEvent_t*) Event_stop->event_backend_ptr;
	//hipEventSynchronize(cuda_event_start);
	//hipEventSynchronize(cuda_event_stop);
	Event_start->sync_barrier();
	Event_stop->sync_barrier();
	hipEvent_t cuda_event_start = *(hipEvent_t*) Event_start->event_backend_ptr;
	hipEvent_t cuda_event_stop = *(hipEvent_t*) Event_stop->event_backend_ptr;
	hipEventElapsedTime(&temp_t, cuda_event_start, cuda_event_stop);
	time_ms = (double) temp_t;
	return time_ms;
}

/*****************************************************/
