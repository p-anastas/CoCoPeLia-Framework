///
/// \author Anastasiadis Petros (panastas@cslab.ece.ntua.gr)
///
/// \brief Some CUDA function calls with added error-checking
///

#include <cstdio>
#include <typeinfo>
#include <float.h>
#include <hiprand.h>

#include "backend_wrappers.hpp"

int Event_num_device[128] = {0};

/*****************************************************/
/// Event Status-related functions

const char* print_event_status(event_status in_status){
	switch(in_status){
		case(UNRECORDED):
			return "UNRECORDED";
		case(RECORDED):
			return "RECORDED";
		case(COMPLETE):
			return "COMPLETE";
		case(CHECKED):
			return "CHECKED";
		case(GHOST):
			return "GHOST";
		default:
			error("print_event_status: Unknown state\n");
	}
}

/*****************************************************/
/// Command queue class functions
CommandQueue::CommandQueue()
{
	cqueue_backend_ptr = malloc(sizeof(hipStream_t));
	hipError_t err = hipStreamCreate((hipStream_t*) cqueue_backend_ptr);
	massert(hipSuccess == err, "CommandQueue::CommandQueue - %s\n", hipGetErrorString(err));
}

void CommandQueue::sync_barrier()
{
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipError_t err = hipStreamSynchronize(stream);
	massert(hipSuccess == err, "CommandQueue::sync_barrier - %s\n", hipGetErrorString(err));
}

void CommandQueue::wait_for_event(Event_p Wevent)
{
	if (Wevent->query_status() == CHECKED) return;
	hipStream_t stream = *((hipStream_t*) cqueue_backend_ptr);
	hipEvent_t cuda_event= *(hipEvent_t*) Wevent->event_backend_ptr;
	hipError_t err = hipStreamWaitEvent(stream, cuda_event, 0); // 0-only parameter = future NVIDIA masterplan?
	massert(hipSuccess == err, "CommandQueue::wait_for_event - %s\n", hipGetErrorString(err));
}

/*****************************************************/
/// Event class functions. TODO: Do status = .. commands need lock?
Event::Event()
{
	event_backend_ptr = malloc(sizeof(hipEvent_t));
	int dev_id;  hipGetDevice(&dev_id);
	Event_num_device[dev_id]++;
	id = Event_num_device[dev_id];
	hipError_t err = hipEventCreate(( hipEvent_t*) event_backend_ptr);
	status = UNRECORDED;
	massert(hipSuccess == err, "Event::Event - %s\n", hipGetErrorString(err));
}

void Event::sync_barrier()
{
	if (status == CHECKED) return;
	else if (status == UNRECORDED){
		warning("Event::sync_barrier: Tried to sync unrecorded event\n");
		return;
	}
	hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
	hipError_t err = hipEventSynchronize(cuda_event);
	if (status == RECORDED) status = COMPLETE;
	massert(hipSuccess == err, "Event::sync_barrier - %s\n", hipGetErrorString(err));
}

void Event::record_to_queue(CQueue_p Rr){
	if (Rr == NULL){
		status = CHECKED;
		return;
	}
	else if (status != UNRECORDED){
		warning("Event::record_to_queue: Recording %s event\n", print_event_status(status));
	}
	hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
	hipStream_t stream = *((hipStream_t*) Rr->cqueue_backend_ptr);
	hipError_t err = hipEventRecord(cuda_event, stream);
	status = RECORDED;
	massert(hipSuccess == err, "Event::record_to_queue - %s\n", hipGetErrorString(err));
}

event_status Event::query_status(){
	if (status == CHECKED) return status;
	hipEvent_t cuda_event= *(hipEvent_t*) event_backend_ptr;
	hipError_t err = hipEventQuery(cuda_event);
	if (err == hipSuccess && (status == UNRECORDED ||  status == COMPLETE)) return status;
	else if (err == hipSuccess && status == RECORDED){ // Event has finished but not synched yet!
		status = COMPLETE;
		return status;
	}
	else if (err == hipErrorNotReady && status == RECORDED) return status;
	else if (err == hipErrorNotReady && status == UNRECORDED){
		// this should not happen in a healthy implementation
		warning("Event::query_status: hipErrorNotReady with status == UNRECORDED should not happen\n");
		status = RECORDED;
		return status;
	}
	else error("Event::query_status - %s, status=%s\n", hipGetErrorString(err), print_event_status(status));
}

void Event::checked(){
	if (status == COMPLETE) status = CHECKED;
	else error("Event::checked(): error event was %s,  not COMPLETE()\n", print_event_status(status));
}

void Event::reset(){
	status = UNRECORDED;
}

/*****************************************************/
/// Event-based timer class functions

Event_timer::Event_timer() {
  Event_start = new Event();
  Event_stop = new Event();
  time_ms = 0;
}

void Event_timer::start_point(CQueue_p start_queue)
{
	Event_start->record_to_queue(start_queue);
	//hipStream_t stream = *((hipStream_t*) start_queue->cqueue_backend_ptr);
	//hipEvent_t cuda_event = *(hipEvent_t*) Event_start->event_backend_ptr;
	//hipEventRecord(cuda_event, stream);
}

void Event_timer::stop_point(CQueue_p stop_queue)
{
	Event_stop->record_to_queue(stop_queue);
	//hipStream_t stream = *((hipStream_t*) stop_queue->cqueue_backend_ptr);
	//hipEvent_t cuda_event = *(hipEvent_t*) Event_stop->event_backend_ptr;
	//hipEventRecord(cuda_event, stream);
}

double Event_timer::sync_get_time()
{
	float temp_t;
	//hipEvent_t cuda_event_start = *(hipEvent_t*) Event_start->event_backend_ptr;
	//hipEvent_t cuda_event_stop = *(hipEvent_t*) Event_stop->event_backend_ptr;
	//hipEventSynchronize(cuda_event_start);
	//hipEventSynchronize(cuda_event_stop);
	Event_start->sync_barrier();
	Event_stop->sync_barrier();
	hipEvent_t cuda_event_start = *(hipEvent_t*) Event_start->event_backend_ptr;
	hipEvent_t cuda_event_stop = *(hipEvent_t*) Event_stop->event_backend_ptr;
	hipEventElapsedTime(&temp_t, cuda_event_start, cuda_event_stop);
	time_ms = (double) temp_t;
	return time_ms;
}

/*****************************************************/
